#include "hip/hip_runtime.h"
#pragma once
#include <stdint.h>
#include "types.cuh"

#define lgWARP 5
#define WARP   (1 << lgWARP)


template <class OP>
__device__ __forceinline__
typename OP::ElTp scanIncWarp(volatile typename OP::ElTp *ptr, uint32_t idx) {
  uint8_t lane = idx & 31;

  if (lane == 0) goto end;
  ptr[idx] = OP::apply(ptr[idx -  1], ptr[idx]);

  if (lane < 2) goto end;
  ptr[idx] = OP::apply(ptr[idx -  2], ptr[idx]);

  if (lane < 4) goto end;
  ptr[idx] = OP::apply(ptr[idx -  4], ptr[idx]);

  if (lane < 8) goto end;
  ptr[idx] = OP::apply(ptr[idx -  8], ptr[idx]);

  if (lane < 16) goto end;
  ptr[idx] = OP::apply(ptr[idx - 16], ptr[idx]);

end:
  return OP::remVolatile(ptr[idx]);
}


template <class OP>
__device__ __forceinline__
typename OP::ElTp scanIncWarp_shfl(volatile typename OP::ElTp *ptr, uint32_t idx) {

  typename OP::ElTp my_val = ptr[idx];

  my_val = __shfl_up_sync((uint8_t)  -1, my_val,  1);
  my_val = __shfl_up_sync((uint8_t)  -2, my_val,  2);
  my_val = __shfl_up_sync((uint8_t)  -4, my_val,  4);
  my_val = __shfl_up_sync((uint8_t)  -8, my_val,  8);
  my_val = __shfl_up_sync((uint8_t) -16, my_val, 16);

  ptr[idx] = my_val;

  return my_val;
}


/*
 * block-level inclusive scan, borrowed from handed-out code for weekly 2
 */
template<class OP>
// __device__ inline typename OP::ElTp
__device__  typename OP::ElTp
scanIncBlock(volatile typename OP::ElTp* ptr, uint32_t idx) {
  uint8_t lane   = idx & (WARP-1);
  uint8_t warpid = idx >> lgWARP;

  // perform warp level scan
  typename OP::ElTp res = scanIncWarp<OP>(ptr, idx);
  __syncthreads();

  // place end-of-warp results in the first warp.
  if (lane == (WARP-1))
    ptr[warpid] = res;

  __syncthreads();

  // re-scan first warp
  if (warpid == 0)
    scanIncWarp<OP>(ptr, idx);

  __syncthreads();

  if (warpid > 0)
    res = OP::apply(ptr[warpid-1], res);

  __syncthreads();

  ptr[idx] = res;
  return res;
}


/*
 *  coalesced copy from global to shared mem, borrowed from handed-out code for weekly 2.
 */
template<class OP, uint8_t CHUNK>
__device__ __forceinline__
void copyFromGlb2ShrMem(uint32_t glb_offs,
                        uint32_t N,
                        const typename OP::ElTp    &ne,
                        typename OP::ElTp          *d_inp,
                        volatile typename OP::ElTp *shmem_inp) {
  #pragma unroll
  for (uint8_t i = 0; i < CHUNK; i++) {

    uint16_t loc_ind = threadIdx.x + blockDim.x * i; 
    uint32_t glb_ind = glb_offs + loc_ind;
    typename OP::ElTp elm = ne;
    if (glb_ind < N)
      elm = d_inp[glb_ind];
    shmem_inp[loc_ind] = elm;
  }
}


/*
 *  coalesced copy from shared to global mem, borrowed from handed-out code for weekly 2.
 */
template<class OP, uint8_t CHUNK>
__device__ __forceinline__
void copyFromShr2GlbMem(uint32_t glb_offs,
                        uint32_t N,
                        typename OP::ElTp          *d_out,
                        volatile typename OP::ElTp *shmem_red) {
  #pragma unroll
  for (uint8_t i = 0; i < CHUNK; i++) {

    uint16_t loc_ind = threadIdx.x + blockDim.x * i;

    uint32_t glb_ind = glb_offs + loc_ind;

    if (glb_ind < N)
      d_out[glb_ind] = shmem_red[loc_ind];
  }
}
