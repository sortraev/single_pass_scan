#include "host.cu"
#include "utils.cu"
#include <stdlib.h>


#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

#ifndef VALIDATION_RUNS
#define VALIDATION_RUNS 2
#endif

int main(int argc, char **argv) {

  int64_t num_blocks_request = -1;
  if (argc == 3)
    num_blocks_request = atoi(argv[2]);
  else if (argc != 2) {
    fprintf(stderr, "Usage: %s <input size> [optional #physical blocks]\n",
            argv[0]);
    return 1;
  }
  uint32_t N = atoi(argv[1]);
  uint32_t alloc_size = N * sizeof(MyInt::ElTp);

  // device input/output memory, and host memory for the GPU kernel result.
  MyInt::ElTp *d_in, *d_out;
  CUDASSERT(hipMalloc(&d_in,  alloc_size));
  CUDASSERT(hipMalloc(&d_out, alloc_size));
  CUDASSERT(hipMemset(d_out, 0, alloc_size));

  // host mem for the GPU kernel result.
  MyInt::ElTp *h_out = (MyInt::ElTp*) calloc(sizeof(MyInt::ElTp), N);
  assert(h_out != NULL);


  // host input/output memory for the reference program.
  MyInt::ElTp *seq_in, *seq_out;
  assert((seq_in  = (MyInt::ElTp*) malloc(alloc_size)) != NULL);
  assert((seq_out = (MyInt::ElTp*) malloc(alloc_size)) != NULL);

  // init input arrays.
  init_array<MyInt, BLOCK_SIZE>(N, d_in);
  CUDASSERT(hipMemcpy(seq_in, d_in, alloc_size, hipMemcpyDeviceToHost));


  // call GPU kernel and copy result to host mem.

  for (int i = 0; i < VALIDATION_RUNS; i++) {
    single_pass_scan
      <Add<MyInt>, BLOCK_SIZE, BLOCK_VIRT>
      (N, d_in, d_out, num_blocks_request,
       i == 0);
  }

  CUDASSERT(hipMemcpy(h_out, d_out, alloc_size, hipMemcpyDeviceToHost));

  // call reference program.
  seq_scan<Add<MyInt> >(N, seq_in, seq_out);

  bool result = validate<MyInt>(N, seq_out, h_out);

  CUDASSERT(hipFree(d_in)); CUDASSERT(hipFree(d_out));
  free(h_out);
  free(seq_in); free(seq_out);

  return result;
}
