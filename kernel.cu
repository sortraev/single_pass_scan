#include "hip/hip_runtime.h"
#pragma once
#include "kernel_extras.cu"

// #if ((BLOCK_SIZE) % 32 != 0)
// #error BLOCK_SIZE must be a multiple of 32.
// #endif
// #if ((MAX_CHUNK <= 0))
// #error MAX_CHUNK must be positive.
// #endif

#define FIRST_IN_BLOCK (!threadIdx.x)
#define FIRST_BLOCK    (!blockIdx)

__device__ uint32_t dyn_gic = 0;     // (dyn)amic (g)lobal (i)ndex (c)ounter

template<class OP, uint8_t CHUNK>
__global__
void spas_kernel(uint32_t           N,             // input size in #elements
                 typename OP::ElTp *d_in,          // scan dis!
                 typename OP::ElTp *d_out,         // store res here!
                 typename OP::ElTp *prefixes,
                 typename OP::ElTp *aggregates,
                 uint8_t           *status_flags,
                 uint32_t           num_logical_blocks
                ) {

  typedef typename OP::ElTp ElTp;
  typedef ValFlg<ElTp> FVpair;

  extern __shared__ uint8_t ext_shmem[];
  uint32_t *blockIdx_shmem = (uint32_t*) ext_shmem;
  ElTp     *shmem        = (ElTp*)     ext_shmem;
  FVpair   *fvp_shmem    = (FVpair*)   ext_shmem;

  bool LAST_IN_BLOCK = threadIdx.x + 1 == blockDim.x;
  ElTp chunk[CHUNK];


#if BLOCK_VIRT
  const uint32_t virt_factor = CEIL_DIV(num_logical_blocks, gridDim.x);
  for (int _ = 0; _ < virt_factor; _++)
#endif
  {
  /*
   * step 1: dynamic block indexing
   */
  if (FIRST_IN_BLOCK) {

    uint32_t tmp = atomicAdd(&dyn_gic, 1);
    *blockIdx_shmem = tmp;               // increment dynamic block index
    status_flags[tmp] = flag_X;
                                         // and publish to the rest of the block
#if !(BLOCK_VIRT)
    // when not using virtualization, simply let the last block reset the
    // counter. this is safe since no more blocks are spawned.
    if (tmp == gridDim.x - 1)
      dyn_gic = 0;
#endif
  }

  __syncthreads();
  uint32_t blockIdx = *blockIdx_shmem; // each thread fetches its dynamic blockIdx and stores it locally

#if BLOCK_VIRT
  if (blockIdx >= num_logical_blocks) {
    // TODO: find out how to safely reset dyn_gic.
    if (FIRST_IN_BLOCK)
      dyn_gic = 0;
    return;
  }
#endif

  /*
   * step 2: each thread copies CHUNK elements from global to shared memory
   */
  uint32_t global_block_offset = blockIdx * blockDim.x * CHUNK;
  copyFromGlb2ShrMem<OP, CHUNK>(global_block_offset, N, OP::ne(), (ElTp*) d_in, shmem);

  __syncthreads();

  /*
   * step 3: each thread copies CHUNK elements from shared mem into own "chunk" array;
   *         performs sequential scan of this and places own result back into shared mem.
   */
  uint16_t shmem_offset = threadIdx.x * CHUNK;

  // copy from shared memory to private chunk.
  #pragma unroll
  for (uint8_t i = 0; i < CHUNK; i++)
    chunk[i] = shmem[shmem_offset + i];


  // perform in-place inclusive scan of chunk and store result in shared memory.
  ElTp acc = OP::ne();
  #pragma unroll
  for (uint8_t i = 0; i < CHUNK; i++)
    chunk[i] = acc = OP::apply(acc, chunk[i]);


  __syncthreads();
  shmem[threadIdx.x] = acc;

  /*
   * step 4: in-place block level scan of shmem. store result in block_aggregate.
   *         (this value only meaningful for last thread in each block)
   */
  ElTp block_aggregate = scanIncBlock<OP>(shmem, threadIdx.x);

  if (LAST_IN_BLOCK) {
    (FIRST_BLOCK ? prefixes : aggregates)[blockIdx] = block_aggregate;
    __threadfence();
    status_flags[blockIdx] = FIRST_BLOCK; // = 1 = flag_P if first block; else = 0 = flag_A>
  }

  __syncthreads();

  ElTp chunk_exc_prefix = OP::ne();
  if (!FIRST_IN_BLOCK)
    chunk_exc_prefix = shmem[threadIdx.x-1];       // extract chunk prefixes before shared mem is reused.


  /*
   * step 6: decoupled lookback to compute exclusive prefix.
   */
  ElTp block_exc_prefix = OP::ne();
  if (!FIRST_BLOCK) {

    if (threadIdx.x < WARP) { // only first warp in block performs lookback

      int32_t lookback_idx = blockIdx + threadIdx.x - WARP;
      while (1) {

        FVpair my_fvp = FVpair(flag_P, OP::ne());

        // choose whether to read an aggregate or prefix depending on the flag
        if (lookback_idx >= 0) {
          my_fvp.f = status_flags[lookback_idx];

          if (my_fvp.f & flag_P)
            my_fvp.v = prefixes[lookback_idx];
          else if (!my_fvp.f)
            my_fvp.v = aggregates[lookback_idx];
        }

        fvp_shmem[threadIdx.x] = my_fvp;

        scanIncWarp<FVpairOP<OP> >(fvp_shmem, threadIdx.x);

        FVpair warp_scan_res = FVpairOP<OP>::remVolatile(fvp_shmem[WARP-1]);

        if (warp_scan_res.f >= flag_X) continue;

        if (FIRST_IN_BLOCK) block_exc_prefix = OP::apply(block_exc_prefix, warp_scan_res.v);

        if (warp_scan_res.f & flag_P) break;

        lookback_idx -= WARP;
      }
    }

    /*
     * step 7: publish block_exc_prefix to rest of block before
     *         letting LAST_IN_BLOCK publish block prefix.
     */
    if (FIRST_IN_BLOCK)
      *shmem = block_exc_prefix;

    __syncthreads();
    block_exc_prefix = *shmem;
  }


  if (!FIRST_BLOCK && LAST_IN_BLOCK) {
    prefixes[blockIdx] = OP::apply(block_exc_prefix, block_aggregate);
    // __threadfence_block();
    __threadfence();
    status_flags[blockIdx] = flag_P;
  }


  /*
   * step 7: mapping the exclusive prefix and copying back to global mem
   */
  ElTp my_prefix = OP::apply(block_exc_prefix, chunk_exc_prefix);

  __syncthreads();

  // map my_prefix over private chunk and write to shared mem
  #pragma unroll
  for (uint8_t i = 0; i < CHUNK; i++)
    shmem[shmem_offset + i] = OP::apply(my_prefix, chunk[i]);

  __syncthreads();

  copyFromShr2GlbMem<OP, CHUNK>(global_block_offset, N, (ElTp*) d_out, shmem);
  }
}
