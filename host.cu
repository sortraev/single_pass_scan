#include "hip/hip_runtime.h"
#include "kernel.cu"
#include "utils.cu"
#include "types.h"
#include "kernel_extras.cu"

template <class OP, uint16_t B, bool do_block_virtualization>
int single_pass_scan(uint32_t           N,
                     typename OP::ElTp *d_in,
                     typename OP::ElTp *d_out,
                     int64_t num_blocks_request = -1
                    ) {

  typedef typename OP::ElTp ElTp;
  typedef ValFlg<ElTp>      FVpair;

  const uint16_t chunk_shmem_bound = MAX_SHMEM / (B * sizeof(ElTp));
  const uint8_t  chunk             = MIN(MAX_CHUNK, chunk_shmem_bound);
  const uint32_t elems_per_block = B * chunk;

  uint32_t num_logical_blocks  = (N + elems_per_block - 1) / elems_per_block;
  uint32_t num_physical_blocks = num_logical_blocks;
  if (do_block_virtualization && num_blocks_request > 0)
    num_physical_blocks = MIN((uint32_t) num_blocks_request,
                              num_logical_blocks);

  ElTp     *aggregates, *prefixes;
  uint8_t  *status_flags;
  CUDASSERT(hipMalloc(&aggregates,   num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&prefixes,     num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&status_flags, num_logical_blocks*sizeof(uint8_t)));
  CUDASSERT(hipMemset(status_flags, flag_X, num_logical_blocks*sizeof(uint8_t)));

  uint32_t shmem_size = max(elems_per_block * sizeof(ElTp),
                            WARP * sizeof(FVpair));

  printf("spas_kernel\n"
         "  block virtualization = %d\n\n"
         "  block size       = %d\n\n"
         "  #logical blocks   = %d\n"
         "  #requested blocks = %d\n"
         "  #physical blocks  = %d\n\n"
         "  chunk      = %d\n"
         "  shmem_size = %d\n"
         "  N = %d\n",
         do_block_virtualization,
         B,
         num_logical_blocks,
         num_blocks_request,
         num_physical_blocks,
         chunk, shmem_size, N);

  spas_kernel<OP, chunk><<<num_physical_blocks, B, shmem_size>>>(
    N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks);

  CUDASSERT(hipFree(aggregates));
  CUDASSERT(hipFree(prefixes));
  CUDASSERT(hipFree(status_flags));

  return 0;
}
