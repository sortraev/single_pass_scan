#include "hip/hip_runtime.h"
#include "kernel.cu"
#include "utils.cu"
#include "types.h"
#include "kernel_extras.cu"

template <class OP, uint16_t B, bool do_block_virtualization>
int single_pass_scan(uint32_t           N,
                     typename OP::ElTp *d_in,
                     typename OP::ElTp *d_out,
                     int64_t num_requested_blocks = -1,
                     bool show_config = false
                    ) {

  typedef typename OP::ElTp ElTp;
  typedef ValFlg<ElTp>      FVpair;

  const uint16_t chunk_shmem_bound = MAX_SHMEM / (B * sizeof(ElTp));
  const uint8_t  chunk             = MIN(MAX_CHUNK, chunk_shmem_bound);
  const uint32_t elems_per_block   = B * chunk;
  uint32_t shmem_size = max(elems_per_block * sizeof(ElTp),
                            WARP * sizeof(FVpair));

  uint32_t num_logical_blocks  = CEIL_DIV(N, elems_per_block);
  uint32_t num_physical_blocks = num_logical_blocks;
  if (do_block_virtualization && num_requested_blocks > 0)
    num_physical_blocks = MIN((uint32_t) num_requested_blocks, num_logical_blocks);
  uint32_t virt_factor = CEIL_DIV(num_logical_blocks, num_physical_blocks);

  uint32_t num_virtblocks = virt_factor * num_physical_blocks;
  uint32_t num_residual_virtblocks = num_virtblocks - num_logical_blocks;

  ElTp     *aggregates, *prefixes;
  uint8_t  *status_flags;
  CUDASSERT(hipMalloc(&aggregates,   num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&prefixes,     num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&status_flags, num_logical_blocks*sizeof(uint8_t)));
  CUDASSERT(hipMemset(status_flags, flag_X, num_logical_blocks*sizeof(uint8_t)));

#if BLOCK_VIRT
  uint32_t *dyn_gic;
  CUDASSERT(hipMalloc(&dyn_gic, sizeof(uint32_t)));
  CUDASSERT(hipMemset(dyn_gic, 0, sizeof(uint32_t)));
#endif

  if (show_config)
    printf("spas_kernel bench\n"
           "  block virt = %d\n\n"

           "  block size        = %d\n\n"

           "  #requested blocks = %d\n"
           "  #logical blocks   = %d\n"
           "  #spawned blocks   = %d\n\n"

           "  virtualization factor = %d\n"
           "  #virtblocks           = %d\n"
           "  #residual_virtblocks  = %d\n\n"

           "  chunk      = %d\n"
           "  shmem_size = %d\n"
           "  N          = %d\n",
           do_block_virtualization,

           B,

           num_requested_blocks,
           num_logical_blocks,
           num_physical_blocks,

           virt_factor,
           num_virtblocks,
           num_residual_virtblocks,
           chunk, shmem_size, N);

  spas_kernel
    <OP, chunk>
    <<<num_physical_blocks, B, shmem_size>>>
#if BLOCK_VIRT
      (N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks, dyn_gic);
  hipMemset(dyn_gic, 0, sizeof(uint32_t));
#else
      (N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks);
#endif

  CUDASSERT(hipFree(aggregates));
  CUDASSERT(hipFree(prefixes));
  CUDASSERT(hipFree(status_flags));

  return 0;
}

// template <class OP>
// int single_pass_scan_no_alloc_aux_arrays(
//     uint32_t N,
//     typename OP::ElTp *d_in,
//     typename OP::ElTp *d_out
// ) {
//   typedef typename OP::ElTp ElTp;
//   typedef ValFlg<ElTp>      FVpair;
//
//   const uint16_t chunk_shmem_bound = MAX_SHMEM / (B * sizeof(ElTp));
//   const uint8_t  chunk             = MIN(MAX_CHUNK, chunk_shmem_bound);
//
//   uint32_t num_logical_blocks  = (N + elems_per_block - 1) / elems_per_block;
//   uint32_t num_physical_blocks = num_logical_blocks;
//   if (do_block_virtualization && num_requested_blocks > 0)
//     num_physical_blocks = MIN((uint32_t) num_requested_blocks,
//                               num_logical_blocks);
//
//   uint32_t array_size = N * sizeof(ElTp);
//
  /*
   * allocate auxiliary arrays
   */
//
//   CUDASSERT(hipMemset(d_out, 0, array_size));
//
//   ElTp     *aggregates, *prefixes;
//   uint8_t  *status_flags;
//   CUDASSERT(hipMalloc(&aggregates,   num_physical_blocks*sizeof(ElTp)));
//   CUDASSERT(hipMalloc(&prefixes,     num_physical_blocks*sizeof(ElTp)));
//   CUDASSERT(hipMalloc(&status_flags, num_physical_blocks*sizeof(uint32_t)));
//   CUDASSERT(hipMemset(status_flags, flag_X, num_physical_blocks*sizeof(uint8_t)));
//   uint32_t shared_mem_size = max(elems_per_block * sizeof(ElTp),
//                                  WARP * sizeof(FVpair));
//
//   // printf("(N, B, CHUNK, num_physical_blocks) == (%d, %d, %d, %d)\n", N, B, chunk, num_physical_blocks);
//   // printf("shared mem per thread: %d\n", shared_mem_size / BLOCK_SIZE);
//   // printf("=========================\n\n");
//
  /*
   *  ================== BENCHMARKING ==================
   */
//     hipEvent_t t_start, t_end;
//     CUDASSERT(hipEventCreate(&t_start)); CUDASSERT(hipEventCreate(&t_end));
//
//     // dry run of kernel
//     spas_kernel<OP, chunk><<<num_physical_blocks, B, shared_mem_size>>>
//       (N, d_in, d_out, prefixes, aggregates, status_flags);
//
//     // invoke SPAS kernel RUNS number of times, measuring total execution time
//     CUDASSERT(hipEventRecord(t_start));
//     for (uint8_t i = 0; i < RUNS; i++) {
//       spas_kernel<OP, chunk><<<num_physical_blocks, B, shared_mem_size>>>
//         (N, d_in, d_out, prefixes, aggregates, status_flags);
//     }
//     CUDASSERT(hipEventRecord(t_end));
//     CUDASSERT(hipEventSynchronize(t_end)); CUDASSERT(hipPeekAtLastError());
//
//
//
//     // get elapsed and report benchmark result
//     float elapsed = get_elapsed(t_start, t_end, RUNS);
//     float GBPerSec = 2*N*sizeof(ElTp)*0.001 / elapsed;
//
//     printf("--  runs in:       %.1lf microseconds\n", elapsed);
//     printf("--  bandwidth:     %.1f GB/sec\n", GBPerSec);
//     // printf("%d, %lf, %d, %d\n", N, GBPerSec, B, chunk);
//   }
//
//
//   CUDASSERT(hipFree(aggregates));
//   CUDASSERT(hipFree(prefixes));
//   CUDASSERT(hipFree(status_flags));
//
//   return 0;
// }
