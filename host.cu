#include "hip/hip_runtime.h"
#include "kernel.cu"
#include "utils.cu"
#include "types.h"
#include "kernel_extras.cu"

template <class OP, uint16_t B, bool do_block_virtualization>
int single_pass_scan(uint32_t           N,
                     typename OP::ElTp *d_in,
                     typename OP::ElTp *d_out,
                     int64_t num_blocks_request = -1
                    ) {

  typedef typename OP::ElTp ElTp;
  typedef ValFlg<ElTp>      FVpair;

  const uint16_t chunk_shmem_bound = MAX_SHMEM / (B * sizeof(ElTp));
  const uint8_t  chunk             = MIN(MAX_CHUNK, chunk_shmem_bound);
  const uint32_t elems_per_block   = B * chunk;
  uint32_t shmem_size = max(elems_per_block * sizeof(ElTp),
                            WARP * sizeof(FVpair));

  uint32_t num_logical_blocks  = CEIL_DIV(N, elems_per_block);
  uint32_t num_physical_blocks = num_logical_blocks;
  if (do_block_virtualization && num_blocks_request > 0)
    num_physical_blocks = MIN((uint32_t) num_blocks_request, num_logical_blocks);
  uint32_t virt_factor = CEIL_DIV(num_logical_blocks, num_physical_blocks);

  ElTp     *aggregates, *prefixes;
  uint8_t  *status_flags;
  CUDASSERT(hipMalloc(&aggregates,   num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&prefixes,     num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&status_flags, num_logical_blocks*sizeof(uint8_t)));
  CUDASSERT(hipMemset(status_flags, flag_X, num_logical_blocks*sizeof(uint8_t)));


  printf("spas_kernel bench\n"
         "  block virt = %d\n\n"
         "  block size        = %d\n\n"
         "  #logical blocks   = %d\n"
         "  #requested blocks = %d\n"
         "  #physical blocks  = %d\n"
         "  virt factor       = %d\n\n"
         "  chunk      = %d\n"
         "  shmem_size = %d\n"
         "  N          = %d\n",
         do_block_virtualization,
         B,
         num_logical_blocks,
         num_blocks_request,
         num_physical_blocks,
         virt_factor,
         chunk, shmem_size, N);

  spas_kernel<OP, chunk><<<num_physical_blocks, B, shmem_size>>>(
    N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks,
    virt_factor);

  CUDASSERT(hipFree(aggregates));
  CUDASSERT(hipFree(prefixes));
  CUDASSERT(hipFree(status_flags));

  return 0;
}

// template <class OP>
// int single_pass_scan_no_alloc_aux_arrays(
//     uint32_t N,
//     typename OP::ElTp *d_in,
//     typename OP::ElTp *d_out
// ) {
//   typedef typename OP::ElTp ElTp;
//   typedef ValFlg<ElTp>      FVpair;
//
//   const uint16_t chunk_shmem_bound = MAX_SHMEM / (B * sizeof(ElTp));
//   const uint8_t  chunk             = MIN(MAX_CHUNK, chunk_shmem_bound);
//
//   uint32_t num_logical_blocks  = (N + elems_per_block - 1) / elems_per_block;
//   uint32_t num_physical_blocks = num_logical_blocks;
//   if (do_block_virtualization && num_blocks_request > 0)
//     num_physical_blocks = MIN((uint32_t) num_blocks_request,
//                               num_logical_blocks);
//
//   uint32_t array_size = N * sizeof(ElTp);
//
  /*
   * allocate auxiliary arrays
   */
//
//   CUDASSERT(hipMemset(d_out, 0, array_size));
//
//   ElTp     *aggregates, *prefixes;
//   uint8_t  *status_flags;
//   CUDASSERT(hipMalloc(&aggregates,   num_physical_blocks*sizeof(ElTp)));
//   CUDASSERT(hipMalloc(&prefixes,     num_physical_blocks*sizeof(ElTp)));
//   CUDASSERT(hipMalloc(&status_flags, num_physical_blocks*sizeof(uint32_t)));
//   CUDASSERT(hipMemset(status_flags, flag_X, num_physical_blocks*sizeof(uint8_t)));
//   uint32_t shared_mem_size = max(elems_per_block * sizeof(ElTp),
//                                  WARP * sizeof(FVpair));
//
//   // printf("(N, B, CHUNK, num_physical_blocks) == (%d, %d, %d, %d)\n", N, B, chunk, num_physical_blocks);
//   // printf("shared mem per thread: %d\n", shared_mem_size / BLOCK_SIZE);
//   // printf("=========================\n\n");
//
  /*
   *  ================== BENCHMARKING ==================
   */
//     hipEvent_t t_start, t_end;
//     CUDASSERT(hipEventCreate(&t_start)); CUDASSERT(hipEventCreate(&t_end));
//
//     // dry run of kernel
//     spas_kernel<OP, chunk><<<num_physical_blocks, B, shared_mem_size>>>
//       (N, d_in, d_out, prefixes, aggregates, status_flags);
//
//     // invoke SPAS kernel RUNS number of times, measuring total execution time
//     CUDASSERT(hipEventRecord(t_start));
//     for (uint8_t i = 0; i < RUNS; i++) {
//       spas_kernel<OP, chunk><<<num_physical_blocks, B, shared_mem_size>>>
//         (N, d_in, d_out, prefixes, aggregates, status_flags);
//     }
//     CUDASSERT(hipEventRecord(t_end));
//     CUDASSERT(hipEventSynchronize(t_end)); CUDASSERT(hipPeekAtLastError());
//
//
//
//     // get elapsed and report benchmark result
//     float elapsed = get_elapsed(t_start, t_end, RUNS);
//     float GBPerSec = 2*N*sizeof(ElTp)*0.001 / elapsed;
//
//     printf("--  runs in:       %.1lf microseconds\n", elapsed);
//     printf("--  bandwidth:     %.1f GB/sec\n", GBPerSec);
//     // printf("%d, %lf, %d, %d\n", N, GBPerSec, B, chunk);
//   }
//
//
//   CUDASSERT(hipFree(aggregates));
//   CUDASSERT(hipFree(prefixes));
//   CUDASSERT(hipFree(status_flags));
//
//   return 0;
// }
