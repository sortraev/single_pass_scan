#pragma once
#include <hip/hip_runtime.h>

#include <stdint.h>
#include <stdio.h>
#include <assert.h>
#include <iostream>


/* if hyperparameters not given as flags to nvcc, set them here */
#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif
#ifndef MAX_CHUNK
#define MAX_CHUNK 9
#endif

#define RUNS      1   /* number of runs performed during benchmarking. */
#define MAX_SHMEM 49152 /* upper bound on shared memory. same for both GPUs, but in the
                           future, we should compute this dynamically for portability.  */

#define MIN(x, y) ((x) < (y) ? x : y) /* since we need a compile-time constant min() function to compute CHUNK */

#define flag_A ((uint8_t) 0)
#define flag_P ((uint8_t) 1)
#define flag_X ((uint8_t) 3)

/*
 * cuda function error wrapper
 */
void CUDASSERT(hipError_t code) {
  if (code != hipSuccess) {
    fprintf(stderr, "Cuda error: %s\n", hipGetErrorString(code));
    exit(code);
  }
}


/*
 * sequential scan implementation used for validation.
 */
template<class OP>
typename OP::ElTp seq_scan(uint32_t           N,
                           typename OP::ElTp *h_in,
                           typename OP::ElTp *h_out) {

  typename OP::ElTp acc = OP::ne();
  for (uint32_t i = 0; i < N; i++)
    h_out[i] = acc = OP::apply(acc, h_in[i]);

  return acc;
}

/*
 * given pointers to two OP::ElTp arrays
 * ref and actual, asserts that they are equal.
 */
template<class OP>
bool validate(uint32_t           N,
              typename OP::ElTp *ref,
              typename OP::ElTp *actual) {

  for (uint32_t i = 0; i < N; i++) {
    if (!OP::equals(ref[i], actual[i])) {

      fprintf(stderr, "\nINVALID!! printing next 10 ...\n"
                      "idx      ref     actual       diff\n");
      for (size_t j = i; j < i + 10; j++)
        fprintf(stderr, "%-8d %-12d %-12d %-10d\n",
                j, ref[j], actual[j], ref[j] - actual[j]);

      return false;
    }
  }

  printf("-- VALID\n");
  return true;
}



/* 
 * kernel to initialize random array in device mem
 */
template <class OP> __global__
void init_device_array(uint32_t           N,
                       typename OP::ElTp *d_in) {
  uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < N)
    d_in[gid] = OP::get_random(gid); // use thread id as seed for computationally cheap "random" numbers
}

/*
 * initializes an OP::ElTp array in device mem
 */
template <class OP, uint16_t B>
void init_array(uint32_t           N,
                typename OP::ElTp *d_in) {

  const uint32_t num_blocks = (N + B - 1) / B;
  init_device_array<OP><<<num_blocks, B>>>(N, d_in);
}

/*
 * slightly more convenient interface for computing elapsed time with cudaEvents
 */
float get_elapsed(hipEvent_t t_start,
                  hipEvent_t t_end,
                  uint32_t    runs) { 
  float elapsed;
  CUDASSERT(hipEventElapsedTime(&elapsed, t_start, t_end));
  elapsed *= 1000 / runs; // convert to microseconds and compute average
  return elapsed;
}


/*
 * for the purpose of comparing with a "realistic" bandwidth number
 */
template <class OP>
__global__ void naiveMemcpy(uint32_t N,
                            typename OP::ElTp *d_in,
                            typename OP::ElTp *d_out) {

  uint32_t gid = blockIdx.x * blockDim.x + threadIdx.x;
  if (gid < N)
    d_out[gid] = d_in[gid];
}

template <class OP, uint16_t B>
int bandwidthMemcpy(const uint32_t N,
                    typename OP::ElTp *d_in,
                    typename OP::ElTp *d_out) {

  typedef typename OP::ElTp ElTp;
  const uint32_t num_blocks = (N + B - 1) / B;
  printf("num_blocks: %d\n", num_blocks);

  // perform dry run of kernel
  naiveMemcpy<OP><<< num_blocks, B >>>(N, d_in, d_out);

  /*
   * run benchmark
   */
  hipEvent_t t_start, t_end;
  CUDASSERT(hipEventCreate(&t_start)); CUDASSERT(hipEventCreate(&t_end));

  CUDASSERT(hipEventRecord(t_start));
  for (int i = 0; i < RUNS; i++) {
    naiveMemcpy<OP><<< num_blocks, B >>>(N, d_in, d_out);
  }
  CUDASSERT(hipEventRecord(t_end));

  /*
   * report elapsed
   */
  CUDASSERT(hipEventSynchronize(t_end));
  float elapsed = get_elapsed(t_start, t_end, RUNS);
  float GBPerSec = 2*N*sizeof(ElTp)*0.001 / elapsed;
  printf("B == %d\n", B);
  printf("--  runs in:       %.1lf microseconds\n", elapsed);
  printf("--  bandwidth:     %.1f GB/sec\n", GBPerSec);
  
  return 0;
}


/*
 * print max_print_len first elements of arr
 * (as long as arr is an int array hehe)
 */
template <class OP>
__device__ __host__
void print_arr(volatile typename OP::ElTp *arr, uint32_t N) {
  int max_print_len = 64;
  N = min(N, max_print_len);
  
  printf("[");
  for (int i = 0; i < N - 1; i++) {
    printf("%d, ", arr[i]);
  }

  if (N > max_print_len)
    printf("...]\n");
  else if (N > 0)
    printf("%d]\n", arr[N - 1]);
  else
    printf("]\n");
}
