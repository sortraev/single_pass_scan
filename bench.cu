#include "hip/hip_runtime.h"
#include "host.cu"
#include "utils.cu"
#include <stdlib.h>

#ifndef BLOCK_SIZE
#define BLOCK_SIZE 256
#endif

#ifndef RUNS
#define RUNS 200
#endif


#define NO_VIRT (!(BLOCK_VIRT))

#if BLOCK_VIRT_FC
#define BLOCK_VIRT 0
#endif

int main(int argc, char **argv) {

  typedef typename MyFloat::ElTp ElTp;
  typedef ValFlg<ElTp> FVpair;

  int64_t num_blocks_request = -1;
  if (argc == 3)
    num_blocks_request = atoi(argv[2]);
  else if (argc != 2) {
    fprintf(stderr, "Usage: %s <input size> [optional #physical blocks]\n",
            argv[0]);
    return 1;
  }

  uint32_t N = atoi(argv[1]);
  uint32_t alloc_size = N * sizeof(ElTp);

  // device input/output memory, and host memory for the GPU kernel result.
  ElTp *d_in, *d_out;
  CUDASSERT(hipMalloc(&d_in,  alloc_size));
  CUDASSERT(hipMalloc(&d_out, alloc_size));

  // init input array.
  init_array<MyFloat, BLOCK_SIZE>(N, d_in);


  // setup kernel parameters.
  const uint16_t chunk_shmem_bound = MAX_SHMEM / (BLOCK_SIZE * sizeof(ElTp));
  const uint8_t  chunk             = MIN(MAX_CHUNK, chunk_shmem_bound);
  const uint32_t elems_per_block   = BLOCK_SIZE * chunk;
  uint32_t shmem_size = max(elems_per_block * sizeof(ElTp),
                            WARP * sizeof(FVpair));

  uint32_t num_logical_blocks  = CEIL_DIV(N, elems_per_block);
  uint32_t num_physical_blocks = num_logical_blocks;
  if (BLOCK_VIRT && num_blocks_request > 0)
    num_physical_blocks = MIN((uint32_t) num_blocks_request, num_logical_blocks);
  uint32_t virt_factor = CEIL_DIV(num_logical_blocks, num_physical_blocks);

  // init auxiliary arrays.
  ElTp     *aggregates, *prefixes;
  uint8_t  *status_flags;
  CUDASSERT(hipMalloc(&aggregates,   num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&prefixes,     num_logical_blocks*sizeof(ElTp)));
  CUDASSERT(hipMalloc(&status_flags, num_logical_blocks*sizeof(uint8_t)));
  CUDASSERT(hipMemset(status_flags, flag_X, num_logical_blocks*sizeof(uint8_t)));

#if BLOCK_VIRT
  uint32_t *dyn_gic;
  CUDASSERT(hipMalloc(&dyn_gic, sizeof(uint32_t)));
  CUDASSERT(hipMemset(dyn_gic, 0, sizeof(uint32_t)));
#endif

  printf("spas_kernel bench\n"
         "  block virt = %d\n\n"
         "  block size        = %d\n\n"
         "  #logical blocks   = %d\n"
         "  #requested blocks = %d\n"
         "  #physical blocks  = %d\n"
         "  virt factor       = %d\n\n"
         "  chunk      = %d\n"
         "  shmem_size = %d\n"
         "  N          = %d\n",
         BLOCK_VIRT,
         BLOCK_SIZE,
         num_logical_blocks,
         num_blocks_request,
         num_physical_blocks,
         virt_factor,
         chunk, shmem_size, N);

    hipEvent_t t_start, t_end;
    CUDASSERT(hipEventCreate(&t_start)); CUDASSERT(hipEventCreate(&t_end));

    // dry run for warmup
    spas_kernel
      <Add<MyFloat>, chunk>
      <<<num_physical_blocks, BLOCK_SIZE, shmem_size>>>
#if BLOCK_VIRT
      (N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks, dyn_gic);
    hipMemset(dyn_gic, 0, sizeof(uint32_t));
#else
      (N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks);
#endif

  CUDASSERT(hipEventRecord(t_start));
#pragma OPTIMIZE OFF
  for (int i = 0; i < RUNS; i++) {
#pragma OPTIMIZE ON

    spas_kernel
      <Add<MyFloat>, chunk>
      <<<num_physical_blocks, BLOCK_SIZE, shmem_size>>>
#if BLOCK_VIRT
      (N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks, dyn_gic);
    hipMemset(dyn_gic, 0, sizeof(uint32_t));
#else
      (N, d_in, d_out, prefixes, aggregates, status_flags, num_logical_blocks);
#endif
  }
  CUDASSERT(hipEventRecord(t_end));
  CUDASSERT(hipEventSynchronize(t_end));
  CUDASSERT(hipPeekAtLastError());


  // get elapsed and report benchmark result
  float elapsed = get_elapsed(t_start, t_end, RUNS);
  float GBPerSec = 2 * N * sizeof(ElTp) * 0.001 / elapsed;

  printf("--  execution time:  %.1lf microseconds\n", elapsed);
  printf("--  bandwidth:       %.1f GB/sec\n", GBPerSec);
  // printf("%d, %lf, %d, %d\n", N, GBPerSec, B, chunk);

  CUDASSERT(hipFree(aggregates));
  CUDASSERT(hipFree(prefixes));
  CUDASSERT(hipFree(status_flags));

  CUDASSERT(hipFree(d_in));
  CUDASSERT(hipFree(d_out));

  return 0;
}
